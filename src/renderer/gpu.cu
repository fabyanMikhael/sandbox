#include "gpu.h"
#include <cstdint>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

uint32_t *gpuAlloc(void) {
    uint32_t *gpu_mem;

    hipError_t err = hipMalloc(&gpu_mem, SCREEN_SIZE * 4);
    if (err != hipSuccess)
        return NULL;

    return gpu_mem;
};

void gpuFree(void *gpu_mem) { hipFree(gpu_mem); }

int gpuBlit(void *src, void *dst) {
    hipError_t err =
        hipMemcpy(dst, src, SCREEN_SIZE * 4, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        return 1;
    return 0;
}
